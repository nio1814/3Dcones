#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>

#include "mp1-util.h"

#ifdef MATLAB_MEX_FILE
#include "mex.h"
#endif

int checkLaunch(const std::string kernel_name)
{
	char msg[128];
	
  hipDeviceSynchronize();
  hipError_t lastErr = hipGetLastError();
  
#ifndef MATLAB_MEX_FILE
  if(lastErr == hipSuccess)
		printf("done with %s kernel\n", kernel_name.c_str());
	else
	{
		printf("error %d '%s' on %s kernel\n", lastErr, hipGetErrorString(lastErr), kernel_name.c_str());
		exit(1);
	}
#else
	if(lastErr == hipSuccess)
		mexPrintf("done with %s kernel\n",kernel_name);
	else
	{
		sprintf(msg, "error %d '%s' on %s kernel\n", lastErr, hipGetErrorString(lastErr), kernel_name);
		mexErrMsgTxt(msg);
	}
#endif
  
  return lastErr;
}

void printExit(char* msg)
{
#ifdef MATLAB_MEX_FILE
	mexErrMsgTxt(msg);
#else
	fprintf(stderr, "%s\n", msg);
	exit(EXIT_FAILURE);
#endif
}


void Timer::start()
{
	int gpuIDcurrent;	//Gpu device called from
	hipGetDevice(&gpuIDcurrent);
	hipSetDevice(0);
	
  hipEventCreate(&this->startEvent);
  hipEventCreate(&this->endEvent);
  hipEventRecord(this->startEvent, 0);
  
	hipSetDevice(gpuIDcurrent);
}


float Timer::stop(const std::string description)
{
	int gpuIDcurrent;	//Gpu device called from
	hipGetDevice(&gpuIDcurrent);
	hipSetDevice(0);
	
  hipEventRecord(this->endEvent, 0);
  hipEventSynchronize(this->endEvent);
  
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, this->startEvent, this->endEvent);
  printf("%s took %.1f ms\n", description.c_str(), elapsed_time);
  hipEventDestroy(this->startEvent);
  hipEventDestroy(this->endEvent);
  
  hipSetDevice(gpuIDcurrent);
  
  return elapsed_time;
}

bool AlmostEqual2sComplement(float A, float B, int maxUlps)
{
    // Make sure maxUlps is non-negative and small enough that the
    // default NAN won't compare as equal to anything.
    // assert(maxUlps > 0 && maxUlps < 4 * 1024 * 1024);
    int aInt = *(int*)&A;
    // Make aInt lexicographically ordered as a twos-complement int
    if (aInt < 0)
        aInt = 0x80000000 - aInt;
    // Make bInt lexicographically ordered as a twos-complement int
    int bInt = *(int*)&B;
    if (bInt < 0)
        bInt = 0x80000000 - bInt;
    int intDiff = abs(aInt - bInt);
    if (intDiff <= maxUlps)
        return true;
    return false;
}

hipMemoryType getPtrLoc(void* ptr)
{
	hipPointerAttribute_t ptrInfo;
	hipMemoryType m;
	
	hipPointerGetAttributes(&ptrInfo, ptr);
	
	m = ptrInfo.memoryType;
	if(ptrInfo.devicePointer!=ptr)
// 		Assume host
		m = hipMemoryTypeHost;
	
	if((m!=hipMemoryTypeHost) && (m!=hipMemoryTypeDevice))
		//assume static memory is on host if no match
		m = hipMemoryTypeHost;
	
	hipGetLastError();
	return m;
}

void findBlockGrid(int npts, int blockSizeIn, size_t *block_size, size_t *grid_size)
{
	*block_size = blockSizeIn;
	*grid_size = npts/(*block_size);
	
// 	Temp fix for oversized grid
	while(*grid_size>65536)
	{
		*block_size *= 2;
		*grid_size = npts/(*block_size);
	}
	
	if(npts%*block_size)
		(*grid_size)++;
	
	return;
}

void findBlockGrid3(int npts, int maxThreadSize, dim3 *block_size, dim3 *grid_size)
{
	int numThreads;
	int numBlocks;
	int deviceID;
	hipDeviceProp_t dp;

	hipGetDevice(&deviceID);
	hipGetDeviceProperties(&dp, deviceID);

	block_size->x = maxThreadSize;
	numBlocks = npts/block_size->x;
	
	// First increase number threads per block
	grid_size->x = numBlocks+1;
	while(grid_size->x>dp.maxGridSize[0] && (block_size->x<(maxThreadSize-32)))
	{
		block_size->x += 32;
		grid_size->x = ceil(npts/(1.0*block_size->x));
	}

	numBlocks = grid_size->x;
	grid_size->y = 1;
	while(grid_size->x>dp.maxGridSize[0])
	{
		grid_size->y++;
		grid_size->x = ceil(numBlocks/(1.0*grid_size->y));
	}
	
	/*
	if(npts%block_size->x)
		grid_size->x++;*/
	
	return;
}

void int3ToInts(int3 d, int *i, int N)
{
	i[0] = d.x;
	if(N>1)
	{
		i[1] = d.y;
		if(N>2)
			i[2] = d.z;
	}

	return;
}

int3 intsToInt3(int *i, int N, int def)
{
	int3 d;
	
	d.x = i[0];
	if(N>1)
		d.y = i[1] ;
	else
		d.y = def;
	
	if(N>2)
		d.z  = i[2];
	else
		d.z = def;

	return d;
}

int3 intsToInt3(int *i, int N)
{
	return intsToInt3(i,N,0);
}

float3 floatsToFloat3(float *i, int N)
{
	float3 d;
	
	d.x = i[0];
	if(N>1)
	{
		d.y = i[1] ;
		if(N>2)
			d.z  =i[2] ;
	}

	return d;
}

int bytes2mb(long m)
{
	m /= 1024;
	m /= 1024;
	
	return m;
}

long prodInt(int* x, int N)
{
	long p;
	int n;
	
	p = x[0];
	
	for(n=1; n<N; n++)
	 p *= x[n];
	
	return p;
}

size_t printAvailMem()
{
	return printAvailMem(0);
}

size_t printAvailMem(int quiet)
{
	size_t memFree, memTotal;
	
	hipMemGetInfo(&memFree, &memTotal);
	if(!quiet)
		printf("%d/%d MB available on gpu\n", bytes2mb(memFree), bytes2mb(memTotal));
	
	return memFree;
}

int setGpu()
{
	int gpuID = 0;
	int numGpu;
	int n;
	size_t memAvail[4];
	size_t memMax = 0;	/* maximum available memory out of all gpus */

	hipGetDeviceCount(&numGpu);

	for(n=0; n<numGpu; n++)
	{
		hipSetDevice(n);
		memAvail[n] = printAvailMem(1); 
		if(memAvail[n]>memMax)
			gpuID = n;
	}

	return gpuID;
}

int findGpuMem(size_t mem)
{
	int gpuID = -1;
	int numGpu;
	int gpuIDOrig;	// Gpu function called from
	size_t memAvail;
	int n;
	
	hipGetDevice(&gpuIDOrig);
	hipGetDeviceCount(&numGpu);
	
	for(n=0; n<numGpu; n++)
	{
		hipSetDevice(n);
		memAvail = printAvailMem(1); 
		if(memAvail>mem)
			gpuID = n;
		if(gpuID>=0)
			break;
	}
	
	hipSetDevice(gpuIDOrig);
	
	
	return gpuID;
}

hipMemcpyKind getCopyFlag(hipMemoryType to, hipMemoryType from)
{
	hipMemcpyKind cf;
	
	if(from==hipMemoryTypeDevice)
		if(to==hipMemoryTypeHost)
			cf = hipMemcpyDeviceToHost;
		else
			cf = hipMemcpyDeviceToDevice;
	else if(from==hipMemoryTypeHost)
		if(to==hipMemoryTypeHost)
			cf = hipMemcpyHostToHost;
		else
			cf = hipMemcpyHostToDevice;
	else
	{
		fprintf(stderr, "Invalid data type %d\n", to);
		exit(EXIT_FAILURE);
	}
	
	return cf;
}

hipMemcpyKind getCopyFlag(void* to, void* from)
{
	hipMemoryType memFrom, memTo;
	memFrom = getPtrLoc(from);
	memTo = getPtrLoc(to);

	return getCopyFlag(memTo, memFrom);
}

int cuFree(void* ptr)
{
	int status = 0;

	hipMemoryType mloc = getPtrLoc(ptr);

	if(mloc==hipMemoryTypeHost)
#ifdef MATLAB_MEX_FILE
		mxFree(ptr);
#else
		hipHostFree(ptr);
#endif
	else if(mloc==hipMemoryTypeDevice)
		hipFree(ptr);
	else
	{
		fprintf(stderr, "Invalid memory location %d\n", mloc);
		status = 1;
	}

	if(!status)
		ptr = NULL;

	return status;
}

int cuMalloc(void** ptr, long size, hipMemoryType mloc)
{
	int status = 0;

	if(mloc==hipMemoryTypeHost)
#ifdef MATLAB_MEX_FILE
		*ptr = mxMalloc(size);
#else
		hipHostMalloc(ptr, size);
#endif
	else if(mloc==hipMemoryTypeDevice)
		hipMalloc(ptr, size);
	else
	{
		fprintf(stderr, "Invalid memory location %d\n", mloc);
		status = 1;
	}

	return status;
}

int cuCalloc(void** ptr, long size, hipMemoryType mloc)
{
	int status;
	
	status = cuMalloc(ptr, size, mloc);

	if(!status)
		status = cuMemset(*ptr, 0, size);

	return status;
}

int cuMemset(void* ptr, int value, size_t size)
{
	hipMemoryType mloc = getPtrLoc(ptr);
	int status = 0;

	if(mloc==hipMemoryTypeHost)
		memset(ptr, value, size);
	else if(mloc==hipMemoryTypeDevice)
		hipMemset(ptr, value, size);
	else
	{
		fprintf(stderr, "Invalid memory location %d\n", mloc);
		status = 1;
	}

	return status;
}

__global__ void testCuComplex(hipComplex* x)
{
	unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

	return;
}

__global__ void testFloat(float * x)
{
	unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

	return;
}
